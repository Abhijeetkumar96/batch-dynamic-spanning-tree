#include "hip/hip_runtime.h"
#include <set>
#include <vector>
#include <string>
#include <chrono>
#include <fstream>
#include <sstream>
#include <iostream>
#include <algorithm>
#include <filesystem>

#include <hip/hip_runtime.h>

#include "connected_components/cc.cuh"

// #define DEBUG

__global__
void initialise(int* parent, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < n) {
        parent[tid] = tid;
    }
}

__global__ 
void hooking(long numEdges, int* original_u, int* original_v, int* d_rep, int* d_flag, int itr_no) 
{
    long tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < numEdges) {
        
        int edge_u = original_u[tid];
        int edge_v = original_v[tid];

        int comp_u = d_rep[edge_u];
        int comp_v = d_rep[edge_v];

        if(comp_u != comp_v) 
        {
            *d_flag = 1;
            int max = (comp_u > comp_v) ? comp_u : comp_v;
            int min = (comp_u < comp_v) ? comp_u : comp_v;

            if(itr_no%2) {
                d_rep[min] = max;
            }
            else { 
                d_rep[max] = min;
            }
        }
    }
}

__global__ 
void short_cutting(int n, int* d_parent) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < n) {
        if(d_parent[tid] != tid) {
            d_parent[tid] = d_parent[d_parent[tid]];
        }
    }   
}

__global__ 
void print_list(int* u, int* v, long numEdges) {
    
    long tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(tid == 0) {
        for(long i = 0; i < numEdges; ++i) {
            printf("edge[%ld]: %d, %d\n", i, u[i], v[i]);
        }
    }
}

std::string cc_get_filename(const std::string& path) {
    return std::filesystem::path(path).stem().string();
}

int cc(int* edge_u, int* edge_v, int numVert, long numEdges, std::string filename) {

    std::vector<int> host_rep(numVert);

    // write the updated the edges to file
    std::vector<int> h_edge_u(numEdges);
    std::vector<int> h_edge_v(numEdges);

    checkCudaError(hipMemcpy(h_edge_u.data(), edge_u, numEdges * sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy back");

    checkCudaError(hipMemcpy(h_edge_v.data(), edge_v, numEdges * sizeof(int), hipMemcpyDeviceToHost),
        "Failed to copy back");

    std::string output_path = "/raid/graphwork/spanning_tree_datasets/bridges_deleted/";
    std::string output_filename = output_path + cc_get_filename(filename) + ".txt";
    std::ofstream outFile(output_filename);

    outFile << numVert << " " << 2 * numEdges << "\n";
    
    for(long i = 0; i < numEdges; ++i) {
        outFile << h_edge_u[i] << " " << h_edge_v[i] << "\n";
        outFile << h_edge_v[i] << " " << h_edge_u[i] << "\n";
    }

    // #ifdef DEBUG
    //     std::cout << "numEdges from cc: " << numEdges << " \n";
    //     print_list<<<1,1>>>(edge_u, edge_v, numEdges);
    //     hipDeviceSynchronize();
    // #endif

    const long numThreads = 1024;
    int numBlocks = (numVert + numThreads - 1) / numThreads;

    int* d_flag;
    checkCudaError(hipMalloc(&d_flag, sizeof(int)), "Unable to allocate flag value");
    auto start = std::chrono::high_resolution_clock::now();
    int* d_rep;
    checkCudaError(hipMalloc(&d_rep, numVert * sizeof(int)), "Unable to allocate rep array");

    initialise<<<numBlocks, numThreads>>>(d_rep, numVert);
    hipError_t err = hipGetLastError();
    checkCudaError(err, "Error in launching initialise kernel");

    int flag = 1;
    int iteration = 0;

    const long numBlocks_hooking = (numEdges + numThreads - 1) / numThreads;
    const long numBlocks_updating_parent = (numVert + numThreads - 1) / numThreads;

    while(flag) {
        flag = 0;
        iteration++;
        checkCudaError(hipMemcpy(d_flag, &flag, sizeof(int),hipMemcpyHostToDevice), "Unable to copy the flag to device");

        hooking<<<numBlocks_hooking, numThreads>>> (numEdges, edge_u, edge_v, d_rep, d_flag, iteration);
        err = hipGetLastError();
        checkCudaError(err, "Error in launching hooking kernel");
        
        // #ifdef DEBUG
        //     hipMemcpy(host_rep.data(), d_rep, numVert * sizeof(int), hipMemcpyDeviceToHost);
        //     // Printing the data
        //     std::cout << "\niteration num : "<< iteration << std::endl;
        //     std::cout << "d_rep : ";
        //     for (int i = 0; i < numVert; i++) {
        //         std::cout << host_rep[i] << " ";
        //     }
        //     std::cout << std::endl;
        // #endif

        for(int i = 0; i < std::ceil(std::log2(numVert)); ++i) {
            short_cutting<<<numBlocks_updating_parent, numThreads>>> (numVert, d_rep);
            err = hipGetLastError();
            checkCudaError(err, "Error in launching short_cutting kernel");
        }

        checkCudaError(hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost), 
            "Unable to copy back flag to host");
    }
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    
    std::cout <<"cc took " << duration << " ms." << std::endl;
    // std::cout <<"Number of iteration: " << iteration << std::endl;
    checkCudaError(hipMemcpy(host_rep.data(), d_rep, numVert * sizeof(int), hipMemcpyDeviceToHost), 
        "Unable to copy back rep array");

    // std::cout << "host_rep array:\n";
    // for(auto i : host_rep)
    //     std::cout << i << " ";

    std::set<int> num_comp(host_rep.begin(), host_rep.end());

    #ifdef DEBUG
        std::cout << "unique comps:\n";
        for(auto i : num_comp)
            std::cout << i << " ";
    #endif

    std::cout <<"\n\nnumComp = " << num_comp.size() << std::endl;

    checkCudaError(hipFree(d_flag), "Failed to free flag");
    checkCudaError(hipFree(d_rep), "Failed to free d_rep");

    return num_comp.size();
}
