#include "hip/hip_runtime.h"
#include "common/cuda_utility.cuh"
#include "dynamic_spanning_tree/euler_tour.cuh"
#include "dynamic_spanning_tree/dynamic_tree_util.cuh"
#include "dynamic_spanning_tree/update_ds.cuh"

#include "hash_table/HashTable.cuh"

// #define DEBUG

using namespace hipcub;

void cpu_pointer_jumping(int* parent, int num_vert) {
    for (int i = 0; i < num_vert; ++i) {
        int root = i;
        // Find the root of the current element
        while (parent[root] != root) {
            root = parent[root];
        }
        // Path compression: update the parent of all elements along the path to point directly to the root
        int current = i;
        while (parent[current] != root) {
            int next = parent[current];
            parent[current] = root;
            current = next;
        }
    }
}

// Constructor
dynamic_tree_manager::dynamic_tree_manager(std::vector<int>& parent, const std::string& delete_filename, const std::vector<uint64_t>& edge_list, int _root) {

    num_vert = parent.size();
    num_edges = edge_list.size();
    root = _root;

    parent_array = new int[num_vert]; // Allocate memory for the array

    // Copy data from the input vector to the newly allocated array
    std::memcpy(parent_array, parent.data(), num_vert * sizeof(int));
    // std::cout << "Reading delete edges file\n";
    read_delete_batch(delete_filename, parent);
    // std::cout << "Reading completed.\n";
    
    // std::cout << "Allocating gpu memory\n";
    mem_alloc(parent, edge_list);
    // std::cout << "Allocation over.\n";

    // std::cout << "Updating data structure\n";
    update_existing_ds();
    std::cout << std::endl;
}

size_t AllocateTempStorage(void** d_temp_storage, long num_items) {
    size_t temp_storage_bytes = 0;
    size_t required_bytes = 0;

    // Determine the temporary storage requirement for DeviceRadixSort::SortPairs
    hipcub::DeviceRadixSort::SortPairs(nullptr, required_bytes, (int*)nullptr, (int*)nullptr, (int*)nullptr, (int*)nullptr, static_cast<int>(num_items));
    temp_storage_bytes = std::max(temp_storage_bytes, required_bytes);

    // Determine the temporary storage requirement for DeviceScan::InclusiveSum
    hipcub::DeviceScan::InclusiveSum(nullptr, required_bytes, (int*)nullptr, (int*)nullptr, static_cast<int>(num_items));
    temp_storage_bytes = std::max(temp_storage_bytes, required_bytes);

    // Determine the temporary storage requirement for DeviceSelect::Flagged
    hipcub::DeviceSelect::Flagged(nullptr, required_bytes, (int*)nullptr, (int*)nullptr, (int*)nullptr, (int*)nullptr, static_cast<int>(num_items));
    temp_storage_bytes = std::max(temp_storage_bytes, required_bytes);

    // Allocate the maximum required temporary storage
    CUDA_CHECK(hipMalloc(d_temp_storage, temp_storage_bytes), "hipMalloc failed for temporary storage for CUB operations");

    return temp_storage_bytes;
}

void dynamic_tree_manager::mem_alloc(const std::vector<int>& parent, const std::vector<uint64_t>& edge_list) {

    size_t size = parent.size() * sizeof(int);
    size_t delete_size = edges_to_delete.size() * sizeof(uint64_t);
    size_t edge_list_size = edge_list.size() * sizeof(uint64_t);
    
    // Allocate device memory

    pHashTable = create_hashtable();

    // allocate temp storage
    AllocateTempStorage(&d_temp_storage, 2 * edge_list.size());

    CUDA_CHECK(hipMalloc(&d_parent, size), "Failed to allocate memory for d_parent");
    CUDA_CHECK(hipMalloc(&new_parent, size), "Failed to allocate memory for d_parent");
    CUDA_CHECK(hipMalloc(&d_org_parent, size), "Failed to allocate memory for d_org_parent");
    CUDA_CHECK(hipMalloc(&d_unique_rep, size), "Failed to allocate memory for d_unique_rep");
    CUDA_CHECK(hipMalloc(&d_rep_map, size), "Failed to allocate memory for d_rep_map");
    CUDA_CHECK(hipMalloc(&d_edges_to_delete, delete_size), "Failed to allocate memory for edges to delete");
    
    // d_edge_list is the original edge_list
    CUDA_CHECK(hipMalloc(&d_edge_list, edge_list_size), "Failed to allocate memory for input edge list");
    
    // d_updated_edge_list is the new edgelist after deleting the edges
    CUDA_CHECK(hipMalloc(&d_updated_edge_list, edge_list_size), "Failed to allocate memory for input edge list");

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_parent, parent.data(), size, hipMemcpyHostToDevice), "Failed to copy d_parent to device");
    CUDA_CHECK(hipMemcpy(d_org_parent, d_parent, size,  hipMemcpyDeviceToDevice), "Failed to copy d_parent to device");
    CUDA_CHECK(hipMemcpy(new_parent, d_parent, size,  hipMemcpyDeviceToDevice), "Failed to copy d_parent to device");
    CUDA_CHECK(hipMemcpy(d_edges_to_delete, edges_to_delete.data(), delete_size, hipMemcpyHostToDevice), "Failed to copy edges to delete to device");
    CUDA_CHECK(hipMemcpy(d_edge_list, edge_list.data(), edge_list_size, hipMemcpyHostToDevice), "Failed to copy edge list to device");
    
    CUDA_CHECK(hipMalloc((void **)&d_super_graph_u, num_edges * sizeof(int)), "Failed to allocate device memory for d_super_graph_u");
    CUDA_CHECK(hipMalloc((void **)&d_super_graph_v, num_edges * sizeof(int)), "Failed to allocate device memory for d_super_graph_v");

    CUDA_CHECK(hipMalloc((void **)&d_new_super_graph_u, num_edges * sizeof(int)), "Failed to allocate device memory for d_new_super_graph_u");
    CUDA_CHECK(hipMalloc((void **)&d_new_super_graph_v, num_edges * sizeof(int)), "Failed to allocate device memory for d_new_super_graph_v");

    CUDA_CHECK(hipMallocManaged((void**)&super_graph_edges, sizeof(int)),   "Failed to allocate d_num_selected_out");
    CUDA_CHECK(hipMalloc((void**)&d_flags, num_edges * sizeof(unsigned char)), "Failed to allocate flag array");
}

void dynamic_tree_manager::read_delete_batch(const std::string& delete_filename, std::vector<int>& parent) {

    std::ifstream inputFile(delete_filename);
    if (!inputFile) {
        std::cerr << "Failed to open file: " << delete_filename << std::endl;
        return;
    }
    
    // n_edges: Number of edges to delete, including both tree and non-tree edges.
    int n_edges;
    inputFile >> n_edges;
    delete_batch_size = n_edges;
    uint32_t u, v;
    edges_to_delete.resize(n_edges);
    
    tree_edge_count = 0;
    
    // std::cout << "Reading " << n_edges << " edges from the file." << std::endl;

    for (int i = 0; i < n_edges; ++i) {
        inputFile >> u >> v;
        if(u > v) {
            // Ensures u is always less than v for consistent edge representation
            std::swap(u, v);
        }

        if(parent_array[u] == v or parent_array[v] == u) {
            tree_edge_count++;

            if(u == parent_array[v]) {
                parent_array[v] = v; // Disconnect the child from its parent

            } else if (v == parent_array[u]) {
                parent_array[u] = u;
            }
        }

        edges_to_delete[i] = ((uint64_t)(u) << 32 | v);
    }
    cpu_pointer_jumping(parent_array, num_vert);

    std::cout << "Number of deleted tree edges: " << tree_edge_count << std::endl;
    
    if(g_verbose) {

        // std::cout << "edges_to_delete array uint64_t:\n";

        // for(auto i : edges_to_delete)
        //     std::cout << i <<" ";
        // std::cout << std::endl;

        // std::cout << "edges_to_delete array:\n";
        // for(const auto &i : edges_to_delete)
        //     std::cout << (i >> 32) << " " << (i & 0xFFFFFFFF) << "\n";
        // std::cout << std::endl;
    }
}

void dynamic_tree_manager::update_existing_ds() {
	update_edgelist(
        d_parent,               // input -- 1
        num_vert,               // input -- 2
        d_edge_list,            // input -- 3
        d_updated_edge_list,    // output -- 4
        num_edges,              // output -- 5
        d_edges_to_delete,      // input -- 6
        delete_batch_size,      // input -- 7
        d_unique_rep,           // output -- 8
        unique_rep_count,       // output -- 9
        root);                  // input -- 10

    // now num_edges contains nonTreeEdges - parent_size - delete_batch count.

    CUDA_CHECK(hipMemcpy(new_parent, d_parent, num_vert * sizeof(int), hipMemcpyDeviceToDevice), 
        "Failed to copy d_parent to device");
}

dynamic_tree_manager::~dynamic_tree_manager() {
    delete[] parent_array;

    hipFree(d_parent);
    hipFree(d_unique_rep);
    hipFree(d_edges_to_delete);
    hipFree(d_edge_list);
    destroy_hashtable(pHashTable);
}

// ====[ End of update ds Code ]====