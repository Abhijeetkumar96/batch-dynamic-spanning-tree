#include "hip/hip_runtime.h"
#include <set>
#include <vector>
#include <string>
#include <chrono>
#include <fstream>
#include <sstream>
#include <iostream>
#include <algorithm>
#include <filesystem>

#include <hip/hip_runtime.h>

#include "connected_components/cc.cuh"

// #define DEBUG

__global__
void initialise(int* parent, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < n) {
        parent[tid] = tid;
    }
}

__global__ 
void hooking(long numEdges, int* original_u, int* original_v, int* d_rep, int* d_flag, int itr_no) 
{
    long tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < numEdges) {
        
        int edge_u = original_u[tid];
        int edge_v = original_v[tid];

        int comp_u = d_rep[edge_u];
        int comp_v = d_rep[edge_v];

        if(comp_u != comp_v) 
        {
            *d_flag = 1;
            int max = (comp_u > comp_v) ? comp_u : comp_v;
            int min = (comp_u < comp_v) ? comp_u : comp_v;

            if(itr_no%2) {
                d_rep[min] = max;
            }
            else { 
                d_rep[max] = min;
            }
        }
    }
}

__global__ 
void short_cutting(int n, int* d_parent) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < n) {
        if(d_parent[tid] != tid) {
            d_parent[tid] = d_parent[d_parent[tid]];
        }
    }   
}

__global__ 
void print_list(int* u, int* v, long numEdges) {
    
    long tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(tid == 0) {
        for(long i = 0; i < numEdges; ++i) {
            printf("edge[%ld]: %d, %d\n", i, u[i], v[i]);
        }
    }
}

std::string cc_get_filename(const std::string& path) {
    return std::filesystem::path(path).stem().string();
}

int cc(int* edge_u, int* edge_v, int numVert, long numEdges, std::string filename) {

    bool _g_verbose = false;
    if(_g_verbose) {
        std::vector<int> host_rep(numVert);
    
        // write the updated the edges to file
        std::vector<int> h_edge_u(numEdges);
        std::vector<int> h_edge_v(numEdges);
    
        checkCudaError(hipMemcpy(h_edge_u.data(), edge_u, numEdges * sizeof(int), hipMemcpyDeviceToHost),
            "Failed to copy back");
    
        checkCudaError(hipMemcpy(h_edge_v.data(), edge_v, numEdges * sizeof(int), hipMemcpyDeviceToHost),
            "Failed to copy back");
    
        std::string output_path = "/raid/graphwork/spanning_tree_datasets/bridges_deleted/";
        std::string output_filename = output_path + cc_get_filename(filename) + ".txt";
        std::ofstream outFile(output_filename);
    
        outFile << numVert << " " << 2 * numEdges << "\n";
        
        for(long i = 0; i < numEdges; ++i) {
            outFile << h_edge_u[i] << " " << h_edge_v[i] << "\n";
            outFile << h_edge_v[i] << " " << h_edge_u[i] << "\n";
        }
    
        // #ifdef DEBUG
        //     std::cout << "numEdges from cc: " << numEdges << " \n";
        //     print_list<<<1,1>>>(edge_u, edge_v, numEdges);
        //     hipDeviceSynchronize();
        // #endif
    }

    const long numThreads = 1024;
    int numBlocks = (numVert + numThreads - 1) / numThreads;

    int* d_flag;
    checkCudaError(hipMalloc(&d_flag, sizeof(int)), "Unable to allocate flag value");

    int* d_rep;
    checkCudaError(hipMalloc(&d_rep, numVert * sizeof(int)), "Unable to allocate rep array");

    initialise<<<numBlocks, numThreads>>>(d_rep, numVert);
    hipError_t err = hipGetLastError();
    checkCudaError(err, "Error in launching initialise kernel");

    int flag = 1;
    int iteration = 0;

    const long numBlocks_hooking = (numEdges + numThreads - 1) / numThreads;
    const long numBlocks_updating_parent = (numVert + numThreads - 1) / numThreads;

    while(flag) {
        flag = 0;
        iteration++;
        checkCudaError(hipMemcpy(d_flag, &flag, sizeof(int),hipMemcpyHostToDevice), "Unable to copy the flag to device");

        hooking<<<numBlocks_hooking, numThreads>>> (numEdges, edge_u, edge_v, d_rep, d_flag, iteration);
        err = hipGetLastError();
        checkCudaError(err, "Error in launching hooking kernel");

        for(int i = 0; i < std::ceil(std::log2(numVert)); ++i) {
            short_cutting<<<numBlocks_updating_parent, numThreads>>> (numVert, d_rep);
            err = hipGetLastError();
            checkCudaError(err, "Error in launching short_cutting kernel");
        }

        checkCudaError(hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost), 
            "Unable to copy back flag to host");
    }

    return 1;
}
