#include "cuda_utility.cuh"
#include "euler_tour.cuh"
#include "dynamic_tree_util.cuh"

using namespace hipcub;

void dynamic_tree_manager::mem_alloc(const std::vector<int>& parent, const std::vector<uint64_t>& edge_list) {

	num_vert = parent.size();
    num_edges = edge_list.size();
    
    size_t size = parent.size() * sizeof(int);
    size_t delete_size = edges_to_delete.size() * sizeof(uint64_t);
    size_t num_edges = edge_list.size() * sizeof(uint64_t);
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_parent, size), "Failed to allocate memory for d_parent");
    CUDA_CHECK(hipMalloc(&d_rep, size), "Failed to allocate memory for d_rep");
    CUDA_CHECK(hipMalloc(&d_unique_rep, size), "Failed to allocate memory for d_unique_rep");
    CUDA_CHECK(hipMalloc(&d_edges_to_delete, delete_size), "Failed to allocate memory for edges to delete");
    CUDA_CHECK(hipMalloc(&d_edge_list, num_edges), "Failed to allocate memory for input edge list");

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_parent, parent.data(), size, hipMemcpyHostToDevice), "Failed to copy d_parent to device");
    CUDA_CHECK(hipMemcpy(d_edges_to_delete, edges_to_delete.data(), delete_size, hipMemcpyHostToDevice), "Failed to copy edges to delete to device");
    CUDA_CHECK(hipMemcpy(d_edge_list, edge_list.data(), num_edges, hipMemcpyHostToDevice), "Failed to copy edge list to device");
    
    // Create a hash table on the device
    pHashTable = create_hashtable();
}

void dynamic_tree_manager::read_delete_batch(const std::string& delete_filename) {

    std::ifstream inputFile(delete_filename);
    if (!inputFile) {
        std::cerr << "Failed to open file: " << delete_filename << std::endl;
        return;
    }
    
    // n_edges: Number of edges to delete, including both tree and non-tree edges.
    int n_edges;
    inputFile >> n_edges;
    delete_batch_size = n_edges;
    uint32_t u, v;
    edges_to_delete.resize(n_edges);
    
    std::cout << "Reading " << n_edges << " edges from the file." << std::endl;

    for (int i = 0; i < n_edges; ++i) {
        inputFile >> u >> v;
        if(u > v) {
            // Ensures u is always less than v for consistent edge representation
            std::swap(u, v);
        }
        edges_to_delete[i] = ((uint64_t)(u) << 32 | v);
    }
}

void dynamic_tree_manager::update_existing_ds() {
	update_existing_ds(
        d_parent, d_rep, num_vert, 
        d_edge_list, num_edges, 
        d_edges_to_delete, delete_batch_size);
}

dynamic_tree_manager::~dynamic_tree_manager() {
    hipFree(d_parent);
    hipFree(d_rep);
    hipFree(d_unique_rep);
    hipFree(d_edges_to_delete);
    hipFree(d_edge_list);
    destroy_hashtable(pHashTable);
}

// ====[ End of update ds Code ]====