#include "hip/hip_runtime.h"
#include "pr_rst/rootedSpanningTreePR.cuh"
#include "pr_rst/grafting.cuh"
#include "pr_rst/reRoot.cuh"
#include "pr_rst/shortcutting.cuh"

__global__ 
void init(int *arr, int *rep, int n) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < n)
	{
		arr[tid] = tid;
		rep[tid] = tid;
	}
}

__global__ 
void init_1(
	int* d_OnPath, 
	int* d_index_ptr, 
	int* d_marked_parent, 
	int* d_winner, 
	int numElements) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        d_OnPath[idx] = 0;
        d_index_ptr[idx] = 0;
        d_marked_parent[idx] = -1;
        d_winner[idx] = -1;
    }
}

void RootedSpanningTree(
	RST_Resource_manager& pr_resources, 
	uint64_t* d_edge_list,
	const int numVert, 
	const int numEdges) {

	int n = numVert;
	int vertices = n;
	int edges = numEdges;

	std::cout << "No. of vertices = " << vertices << std::endl;
	std::cout << "No. of edges = " << edges << std::endl;

	// Update values for pointerJumping
	std::cout << "log2(n) = " << std::log2(n) << std::endl;
	int log_2_size = std::ceil(std::log2(n));
	long long pr_size = std::ceil(n * 1LL * log_2_size);
	std::cout << "pr_size = " << pr_size << std::endl;
	
	long long size = n * 1LL * sizeof(int); // For n vertices

	std::cout << "size: " <<  size << std::endl;

	int *d_winner_ptr = pr_resources.d_winner;
	int *d_ptr = pr_resources.d_ptr;
	int *d_parent_ptr = pr_resources.d_parent_ptr;
	int *d_new_parent_ptr = pr_resources.d_new_parent_ptr; // output
	int *d_pr_arr = pr_resources.d_pr_arr;
	int *d_OnPath = pr_resources.d_OnPath;
	int *d_new_OnPath = pr_resources.d_new_OnPath;
	int *d_marked_parent = pr_resources.d_marked_parent;
	int *d_next = pr_resources.d_next;
	int *d_new_next = pr_resources.d_new_next;
	int *d_index_ptr = pr_resources.d_index_ptr;
	int *d_pr_size_ptr = pr_resources.d_pr_size_ptr;

	// Till here pointerJumping values set up

	int numThreads = 1024;
	int numBlocks_n = (vertices + numThreads - 1) / numThreads;

	auto start = std::chrono::high_resolution_clock::now();

	// Step 1: Initialize rep with vertices themselves
	init<<<numBlocks_n, numThreads>>>(d_ptr, d_parent_ptr, vertices);
	CUDA_CHECK(hipDeviceSynchronize(), "failed to synchronize after init kernel");

	int *d_flag;
	CUDA_CHECK(hipMalloc(&d_flag, sizeof(int)), "failed to allocate memory for d_flag");

	int flag = 1;
	int iter_number = 0;

	while (flag) {
		if(iter_number > 2*log_2_size) {
			std::cerr<<"Iterations exceeded 2*log_2_n : "<<iter_number<<"\n";
			break;
		}

		flag = 0;

		CUDA_CHECK(hipMemcpy(d_flag, &flag, sizeof(int), hipMemcpyHostToDevice), "Failed to copy back d_flag");

		// init_1<<<numBlocks_n, numThreads>>>(d_OnPath, d_index_ptr, d_marked_parent, d_winner_ptr, vertices);

		hipMemset(d_OnPath, 		 0, size);
		hipMemset(d_index_ptr,		 0, size);
		hipMemset(d_marked_parent,	-1, size);
		hipMemset(d_winner_ptr,	-1, size);
		
		//thrust::fill is better optimized than hipMemset
		// thrust::fill(d_winner ,d_winner.end(), -1);

		CUDA_CHECK(hipDeviceSynchronize(), "failed to synchronize after init_1 kernel");

		//Step 2: Graft
		Graft(vertices, edges, d_edge_list, d_ptr, d_winner_ptr, d_marked_parent, d_OnPath, d_flag);

		// Step 3: ReRoot
		ReRoot(vertices, edges, log_2_size, iter_number, d_OnPath, d_new_OnPath , d_pr_arr, d_parent_ptr, d_new_parent_ptr, d_index_ptr, d_pr_size_ptr, d_marked_parent, d_ptr);

		CUDA_CHECK(hipMemcpy(d_next, d_parent_ptr, size, hipMemcpyDeviceToDevice), "Failed to copy d_next");

		// Step 4: Shortcutting
		CUDA_CHECK(hipMemset(d_pr_size_ptr, 0, size), " ");
		CUDA_CHECK(hipMemset(d_pr_arr, -1, pr_size), " ");

		Shortcut(vertices, edges, log_2_size, d_next, d_new_next, d_pr_arr, d_ptr, d_pr_size_ptr);	

		iter_number++;
		CUDA_CHECK(hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost), "Failed to copy back d_flag");
	}

	// return d_new_parent_ptr;
}
