#include "hip/hip_runtime.h"
#include "PR-RST/rootedSpanningTreePR.cuh"
#include "PR-RST/grafting.cuh"
#include "PR-RST/reRoot.cuh"
#include "PR-RST/pr_rst_util.cuh"
#include "PR-RST/shortcutting.cuh"

#include "common/cuda_utility.cuh"

__global__ 
void init(int *arr, int *rep, int n) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < n)
	{
		arr[tid] = tid;
		rep[tid] = tid;
	}
}

__global__ 
void init_arrays(int* d_OnPath, int* d_index_ptr, int* d_marked_parent, int* d_winner_ptr, size_t numElements) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        d_OnPath[idx] = 0;
        d_index_ptr[idx] = 0;

        d_marked_parent[idx] = -1;
        d_winner_ptr[idx] = -1;
    }
}


void RootedSpanningTree(uint64_t* d_edgelist, const int numVert, const int numEdges) {

	int n = numVert;
	int vertices = n;
	int edges = numEdges;

	std::cout << "No. of vertices = " << vertices << std::endl;
	std::cout << "No. of edges = " << edges << std::endl;

	// Update values for pointerJumping
	std::cout << "log2(n) = " << std::log2(n) << std::endl;
	int log_2_size = std::ceil(std::log2(n));
	long long pr_size = std::ceil(n * 1LL * log_2_size);
	std::cout << "pr_size = " << pr_size << std::endl;
	
	long long size = n * 1LL * sizeof(int); // For n vertices

	std::cout << "size: " <<  size << std::endl;

	int *d_winner_ptr;
	int *d_ptr;
	int *d_parent_ptr;
	int *d_new_parent_ptr;
	int *d_pr_arr;
	int *d_label;
	int *d_OnPath;
	int *d_new_OnPath;
	int *d_rep;
	int *d_marked_parent;
	int *d_next;
	int *d_new_next;
	int *d_index_ptr;
	int *d_pr_size_ptr;
	int *d_flag;

	CUDA_CHECK(hipMalloc(&d_flag, sizeof(int)), 						"Failed to allocate memory for d_flag");
	CUDA_CHECK(hipMalloc((void**)&d_winner_ptr, n * sizeof(int)), 		"Failed to allocate memory for d_winner_ptr");
	CUDA_CHECK(hipMalloc((void**)&d_ptr, size), 						"Failed to allocate memory for d_ptr");
	CUDA_CHECK(hipMalloc((void**)&d_parent_ptr, size), 				"Failed to allocate memory for d_parent_ptr");
	CUDA_CHECK(hipMalloc((void**)&d_new_parent_ptr, size), 			"Failed to allocate memory for d_new_parent_ptr");
	CUDA_CHECK(hipMalloc((void**)&d_pr_arr, sizeof(int) * pr_size), 	"Failed to allocate memory for d_pr_arr");
	CUDA_CHECK(hipMalloc((void**)&d_label, size), 						"Failed to allocate memory for d_label");
	CUDA_CHECK(hipMalloc((void**)&d_rep, size), 						"Failed to allocate memory for d_rep");
	CUDA_CHECK(hipMalloc((void**)&d_OnPath, size), 					"Failed to allocate memory for d_OnPath");
	CUDA_CHECK(hipMalloc((void**)&d_new_OnPath, size), 				"Failed to allocate memory for d_new_OnPath");
	CUDA_CHECK(hipMalloc((void**)&d_marked_parent, size), 				"Failed to allocate memory for d_marked_parent");
	CUDA_CHECK(hipMalloc((void**)&d_next, size), 						"Failed to allocate memory for d_next");
	CUDA_CHECK(hipMalloc((void**)&d_new_next, size), 					"Failed to allocate memory for d_new_next");
	CUDA_CHECK(hipMalloc((void**)&d_index_ptr, size), 					"Failed to allocate memory for d_index_ptr");
	CUDA_CHECK(hipMalloc((void**)&d_pr_size_ptr, sizeof(int)), 		"Failed to allocate memory for d_pr_size_ptr");

	// Till here pointerJumping values set up

	int numThreads = 1024;
	int numBlocks_n = (vertices + numThreads - 1) / numThreads;

	auto start = std::chrono::high_resolution_clock::now();

	// Step 1: Initialize rep with vertices themselves
	init<<<numBlocks_n, numThreads>>>(d_ptr, d_parent_ptr, vertices);
	hipDeviceSynchronize();

	int flag = 1;
	int iter_number = 0;

	while (flag) {
		if(iter_number > 2*log_2_size) {
			std::cerr<<"Iterations exceeded 2*log_2_n : "<<iter_number<<"\n";
			break;
		}

		flag = 0;

		CUDA_CHECK(hipMemcpy(d_flag, &flag, sizeof(int), hipMemcpyHostToDevice), 	"Failed to copy flag to device");
	
		int threadsPerBlock = 1024;
		size_t blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
		init_arrays<<<blocksPerGrid, threadsPerBlock>>>(d_OnPath, d_index_ptr, d_marked_parent, d_winner_ptr, n);
		CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize after init_arrays kernel");

		//Step 2: Graft
		Graft(vertices, edges, d_edgelist, d_ptr, d_winner_ptr, d_marked_parent, d_OnPath, d_flag);
		ReRoot(vertices, edges, log_2_size, iter_number, d_OnPath, d_new_OnPath , d_pr_arr, d_parent_ptr, d_new_parent_ptr, d_index_ptr, d_pr_size_ptr, d_marked_parent, d_ptr);
		hipMemcpy(d_next, d_parent_ptr, size, hipMemcpyDeviceToDevice);

		// Step 4: Shortcutting
		hipMemset(d_pr_size_ptr, 0, size);
		hipMemset(d_pr_arr, -1, pr_size);

		Shortcut(vertices, edges, log_2_size, d_next, d_new_next, d_pr_arr, d_ptr, d_pr_size_ptr);	
		
		iter_number++;
		hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
	}
	
	std::vector<int> h_parent(n), h_rep(n);
	hipMemcpy(h_parent.data(), d_parent_ptr, n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_rep.data(), d_ptr, n*sizeof(int), hipMemcpyDeviceToHost);
	
	std::cout << "parent array : \n";

	int j = 0;
	for (auto i : h_parent)
		std::cout << "parent[" << j++ << "] = " << i << std::endl;
	std::cout << std::endl;

	std::cout<<"Parent before exiting module \n\n";
	for(auto i : h_parent){
		std::cout<<i<<" ";
	}

	std::cout<<std::endl;

	hipFree(d_OnPath);
}
