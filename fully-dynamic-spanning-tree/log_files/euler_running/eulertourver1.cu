#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>
#include "headers/eulerTour.h"
#include "headers/mytimer.h"

using namespace std;

__device__ int cuAbs(int i) { return i < 0 ? -i : i; }

__global__ void copy_to_device(int* edges_from_input , int* edges_to_input , int* edges_from , int* edges_to , int N){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < N-1){
        edges_from[thid + N - 1] = edges_to[thid] = edges_to_input[thid];
        edges_to[thid + N - 1] = edges_from[thid] = edges_from_input[thid];
    }
}

__global__ void init_index(int* index , int E){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < E){
        index[i] = i;
    }
}

__global__ void init_next(int* next , int E){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < E){
        next[i] = -1;
    }
}

__global__ void fill_nextandfirst(int* edges_from , int* edges_to , int* index , int* next , int* first , int E){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < E){
        int f = edges_from[index[thid]];
        //int t = edges_to[index[thid]];

        if (thid == 0) {
          first[f] = index[thid];
          return;
        }

        int pf = edges_from[index[thid - 1]];
        //int pt = edges_to[index[thid - 1]];

        if (f != pf) {
          first[f] = index[thid];
        } else {
          next[index[thid - 1]] = index[thid];
        } 
    }
}

__global__ void fill_succ(int* edges_from , int* next , int* first , int* succ , int E){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < E){
        int revEdge = (thid + E / 2) % E;
        if (next[revEdge] == -1) {
          succ[thid] = first[edges_from[revEdge]];
        } else {
          succ[thid] = next[revEdge];
        }
    }
}


__global__ void init_devNext(int* devNext , int* devNextSrc , int N , int head){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < N){
      devNext[i] = devNextSrc[i];
      if (devNextSrc[i] == head)
        devNext[i] = -1;
  }
}

__global__ void init_devRank(int* devRank , int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < N){
      devRank[i] = 0;
  }
}


__global__ void split(int* devNext, int* devSublistHead, int N, int s , int head){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < s){
      hiprandState state;
      hiprand_init(123, i, 0, &state);

      int p = i * (N / s);
      int q = min(p + N / s, N);

      int splitter;
      do {
        splitter = (cuAbs(hiprand(&state)) % (q - p)) + p;
      } while (devNext[splitter] == -1);

      devSublistHead[i + 1] = devNext[splitter];
      devNext[splitter] = -i - 2; // To avoid confusion with -1

      if (i == 0) {
        devSublistHead[0] = head;
      }
  }
}

__global__ void updateall(int* devNext, int* devRank, int* devSublistHead, int* devSublistId, int* devLast, int* devSum , int s){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < s + 1){
      int current = devSublistHead[thid];
      int counter = 0;

      while (current >= 0) {
        
        devRank[current] = counter++;

        int n = devNext[current];

        if (n < 0) {
          devSum[thid] = counter;
          devLast[thid] = current;
        }

        devSublistId[current] = thid;
        current = n;
      }
  }
}


__global__ void kernelf(int head , int s, int* devNext , int* devLast , int* devSum){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid == 0){
      int tmpSum = 0;
      int current = head;
      int currentSublist = 0;
      for (int i = 0; i <= s; i++) {
        tmpSum += devSum[currentSublist];
        devSum[currentSublist] = tmpSum - devSum[currentSublist];

        current = devLast[currentSublist];
        currentSublist = -devNext[current] - 1;
      }
  }
}



__global__ void kernelf1(int* devSum , int* devRank , int* devSublistId , int N){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < N){
        int sublistId = devSublistId[thid];
        devRank[thid] += devSum[sublistId];
    }
}

__global__ void fill_ness(int* d_parent , int* edges_from_input , int* edges_to_input , int N , int root){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < N){
      if (thid == root)return;
    int afterRoot = thid > root;
    edges_from_input[thid - afterRoot] = thid;
    edges_to_input[thid - afterRoot] = d_parent[thid];
    }
}

__global__ void getMET(int* rank_to_output , int* firstOccurrence , int* lastOccurrence , int* mod_euler_tour , int N , int* edges_from , int* edges_to , int* d_level , int root){
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid < 2*N-2){
      int u = edges_from[thid];
      int v = edges_to[thid];
      if(d_level[u] < d_level[v]){
          firstOccurrence[v] = rank_to_output[thid]+1;
          mod_euler_tour[rank_to_output[thid]+1] = 1;
      }
      else{
          lastOccurrence[u] = rank_to_output[thid]+1;
          mod_euler_tour[rank_to_output[thid]+1] = 0;
      }
      
      if(thid==0){
        mod_euler_tour[0] = 1;
        mod_euler_tour[2*N-1] = 0;
        firstOccurrence[root] = 0;
        lastOccurrence[root] = 2*N-1;
      }

  }
}


void cuda_list_rank(int N, int head, int *devNextSrc, int *devRank){

  mytimer mt3{};

  int s;
  if (N >= 100000) {
    s = sqrt(N) * 1.6; // Based on experimental results for GTX 980.
  } else
    s = N / 100;
  if (s == 0) s = 1;

  int *devNext;
  hipMalloc((void **)&devNext, sizeof(int) * (N));
  
  int num_threads = 1024;
  int num_blocks = (N + 1024 - 1) / 1024;
  init_devNext<<<num_blocks , num_threads>>>(devNext  , devNextSrc , N , head);
  hipDeviceSynchronize();

  init_devRank<<<num_blocks , num_threads>>>(devRank , N);
  hipDeviceSynchronize();

  int *devSum;
  hipMalloc((void **)&devSum, sizeof(int) * (s + 1));
  int *devSublistHead;
  hipMalloc((void **)&devSublistHead, sizeof(int) * (s + 1));
  int *devSublistId;
  hipMalloc((void **)&devSublistId, sizeof(int) * N);
  int *devLast;
  hipMalloc((void **)&devLast, sizeof(int) * (s + 1));

  mt3.timetaken_reset("alloc" , 0 );

  num_blocks = (s + 1024 - 1) / 1024;
  split<<<num_blocks , num_threads>>>(devNext, devSublistHead, N, s , head);
  hipDeviceSynchronize();


  num_blocks = (s + 1 + 1024 - 1) / 1024;
  updateall<<<num_blocks , num_threads>>>(devNext, devRank, devSublistHead, devSublistId, devLast, devSum , s);
  hipDeviceSynchronize();

  kernelf<<<1,1>>>(head , s, devNext , devLast , devSum);
  hipDeviceSynchronize();


  num_blocks = (N + 1024 - 1) / 1024;
  kernelf1<<<num_blocks , num_threads>>>(devSum , devRank ,devSublistId , N);
  hipDeviceSynchronize();

  mt3.timetaken_reset("step 6 : list_rank" , 1 );

  hipFree(devNext);
  hipFree(devSum);
  hipFree(devSublistHead);
  hipFree(devSublistId);
  hipFree(devLast);

}



void cuda_euler_tour(int N , int root , int* edges_from_input, int* edges_to_input , int* rank_to_output , int* firstOccurrence , int* lastOccurrence , int* mod_euler_tour , int* d_level){


    mytimer mt2{};

  void *d_temp_storage1 = NULL;
  size_t temp_storage_bytes1 = 0;
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage1, temp_storage_bytes1, mod_euler_tour, mod_euler_tour, 2*N);
  hipMalloc(&d_temp_storage1, temp_storage_bytes1);

    int E = N*2 - 2;

    int *edges_to;
    int *edges_to_copy;
    int *edges_from;
    int *edges_from_copy;
    hipMalloc((void **)&edges_to, sizeof(int) * E);
    hipMalloc((void **)&edges_from, sizeof(int) * E);
    hipMalloc((void **)&edges_to_copy, sizeof(int) * E);
    hipMalloc((void **)&edges_from_copy, sizeof(int) * E);
    int *index;
    hipMalloc((void **)&index, sizeof(int) * E);
    int *index1;
    hipMalloc((void **)&index1, sizeof(int) * E);
    int *index2;
    hipMalloc((void **)&index2, sizeof(int) * E);

    int num_threads = 1024;
    int num_blocks = (N-1 + num_threads - 1) / num_threads;
    copy_to_device<<<num_blocks , num_threads>>>(edges_from_input , edges_to_input , edges_from , edges_to , N);
    hipDeviceSynchronize();

    num_threads = 1024;
    num_blocks = (E + num_threads - 1) / num_threads;
    init_index<<<num_blocks , num_threads>>>(index , E);
    hipDeviceSynchronize();

    int *h_index1;
    hipHostMalloc((void **)&h_index1, sizeof(int) * E);
    hipMemcpy(h_index1, index, sizeof(int) * E, hipMemcpyDeviceToHost);

    
    int *next;
    hipMalloc((void **)&next, sizeof(int) * E);
    int* succ;
    hipMalloc((void **)&succ, sizeof(int) * E);
    int* head;
    hipHostMalloc((void **)&head, sizeof(int) * 1);
    int* first;
    hipMalloc((void **)&first, sizeof(int) * N);


    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, edges_to , edges_to_copy , index , index ,E);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    mt2.timetaken_reset("step 1 : sort" , 0 );

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, edges_to , edges_to_copy , index , index1 ,E);
    
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, edges_to , edges_to_copy , edges_from , edges_from_copy ,E);

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, edges_from_copy , edges_to_copy , index1 , index2 , E);

    

    hipFree(d_temp_storage);
    hipFree(index);
    hipFree(index1);
    hipFree(edges_from_copy);
    hipFree(edges_to_copy);



    init_next<<<num_blocks , num_threads>>>(next , E);
    hipDeviceSynchronize();

    

    fill_nextandfirst<<<num_blocks , num_threads>>>(edges_from , edges_to , index2 , next , first , E);
    hipDeviceSynchronize();

    
    hipFree(index2);

    

    fill_succ<<<num_blocks , num_threads>>>(edges_from , next , first , succ , E);
    hipDeviceSynchronize();

    mt2.timetaken_reset("step 3 : fill_succ" , 1 );

    hipMemcpy(head, first + root, sizeof(int), hipMemcpyDeviceToHost);

    
    hipFree(next);
    hipFree(first);

    cuda_list_rank(E , *head , succ , rank_to_output);

    num_blocks = (2*N - 2 + 1024 - 1) / 1024;

    mt2.timetaken_reset("step 4 : list_rank" , 0 );

    getMET<<<num_blocks , num_threads>>>(rank_to_output , firstOccurrence , lastOccurrence , mod_euler_tour , N , edges_from , edges_to , d_level , root);
    hipDeviceSynchronize();

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage1, temp_storage_bytes1, mod_euler_tour, mod_euler_tour , 2*N);

    mt2.timetaken_reset("step 5 : getMET" , 1 );
    
    hipFree(succ);
    hipFree(edges_to);
    hipFree(edges_from);
}


void getModifiedEulerTour(int *d_parent , int* d_level , int n , int* mod_euler_tour , int* firstOccurrence , int*  lastOccurrence , int root){

  mytimer mt1{};
  
  int* edges_from_input;
  int* edges_to_input;
  hipMalloc((void **)&edges_from_input, sizeof(int) * (n-1));
  hipMalloc((void **)&edges_to_input, sizeof(int) * (n-1));
  int* rank_to_output;
  hipMalloc((void **)&rank_to_output, sizeof(int) * n*2);


  int num_threads = 1024;
  int num_blocks = (n + num_threads - 1) / num_threads;

  mt1.timetaken_reset("alloc" , 0 );

  fill_ness<<<num_blocks , num_threads>>>(d_parent , edges_from_input , edges_to_input , n , root);
  hipDeviceSynchronize();

  mt1.timetaken_reset("step 1 : fill_ness " , 1 );

  cuda_euler_tour(n , root , edges_from_input , edges_to_input , rank_to_output , firstOccurrence , lastOccurrence , mod_euler_tour , d_level);

  hipFree(edges_from_input);
  hipFree(edges_to_input);
  hipFree(rank_to_output);

}





// int main(){
  
//   freopen("amazon_parent.txt", "r", stdin);
//   int n;
//   cin>>n;
//   int* parent;
//   hipHostMalloc((void **)&parent, sizeof(int) * n);
//   int* level;
//   hipHostMalloc((void **)&level, sizeof(int) * n);
//   for(int i=0;i<n;i++){
//     cin>>parent[i];
//   }

//   ifstream file("output_valid_amazon.txt");
//   for(int i=0;i<n;i++){
//     file>>level[i];
//   }
//   file.close();

//   int* d_level;
//   hipMalloc((void **)&d_level, sizeof(int) * n);
//   hipMemcpy(d_level, level, sizeof(int) * n, hipMemcpyHostToDevice);

//   int* d_parent;
//   hipMalloc((void **)&d_parent, sizeof(int) * n);
//   hipMemcpy(d_parent, parent, sizeof(int) * n, hipMemcpyHostToDevice);

//   int* d_mod_euler_tour;
//   hipMalloc((void **)&d_mod_euler_tour, sizeof(int) * n*2);

//   int* d_firstOccurrence;
//   hipMalloc((void **)&d_firstOccurrence, sizeof(int) * n);

//   int* d_lastOccurrence;
//   hipMalloc((void **)&d_lastOccurrence, sizeof(int) * n);

//   getModifiedEulerTour(d_parent , d_level , n , d_mod_euler_tour , d_firstOccurrence , d_lastOccurrence , 0);

    
//   return 0;

// }