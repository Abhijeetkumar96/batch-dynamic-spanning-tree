#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "common/Timer.hpp"
#include "common/cuda_utility.cuh"

#include "dynamic_spanning_tree/dynamic_tree.cuh"

#include "path_reversal/path_reversal.cuh"
#include "PR-RST/reversePaths.cuh"

__global__
void reverse_new_parents_(
	int* edge_u, 
	int* parent_u, 
	int* new_parent, 
	int h_size) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < h_size) {
		// printf("[%d,%d]\n",edge_u[tid],parent_u[tid]);
		new_parent[edge_u[tid]] = parent_u[tid];
	}	
}

void path_reversal_PR(
	dynamic_tree_manager& tree_ds, 
	REP_EDGES& rep_edge_mag, 
	thrust::device_vector<int> &onPath,
	thrust::device_vector<int> &pr_arr,
	thrust::device_vector<int> &pr_arr_size,
	int log_2_size) {

	int num_vert  = tree_ds.num_vert;
	int num_edges = tree_ds.num_edges;
    int* edge_u   = rep_edge_mag.d_edge_u;
    int* parent_u = rep_edge_mag.d_parent_u;
	
	//n == uniqueRep array size

	std::cout << "Executing path_reversal:\n";

	thrust::device_vector <int> onPathCpy(num_vert);
	thrust::device_vector <int> parent_pr_tmp(num_vert);
	thrust::device_vector <int> us1(num_vert);

	int numThreads = 1024;

	auto start = std::chrono::high_resolution_clock::now();

	int p_size = tree_ds.num_vert;
	int* new_parent = tree_ds.new_parent;

    ReversePaths(num_vert, num_edges, log_2_size, 
        thrust::raw_pointer_cast(onPath.data()),
        thrust::raw_pointer_cast(onPathCpy.data()),
        thrust::raw_pointer_cast(pr_arr.data()),
        new_parent,          // changes reflected
        thrust::raw_pointer_cast(parent_pr_tmp.data()),
        thrust::raw_pointer_cast(us1.data()),
        thrust::raw_pointer_cast(pr_arr_size.data())   
    );

    // if(g_verbose) {
    	// std::cout<<"Parent after: ";
		// for(auto i : n_parent)
		// {
		// 	std::cout<<i<<" ";
		// }
		// std::cout<<"\n";
    // }

	// h_size is super_graph parent array size
    int h_size = rep_edge_mag.num_vert;
    int numBlocks = (h_size + numThreads - 1) / numThreads;

	reverse_new_parents_<<<numBlocks, numThreads>>>(
		edge_u, 
        parent_u,
        new_parent, 
        h_size);

	CUDA_CHECK(hipDeviceSynchronize(), 
		"Failed to synchronize after update_parent_kernel");
	
	auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration<double, std::milli>(stop - start).count();

    add_function_time("Path Reversal", duration);

	// g_verbose = false;

	if(g_verbose) {
		std::cout << "New parent array:\n";
		std::vector<int> h_new_parent(p_size);
		CUDA_CHECK(hipMemcpy(h_new_parent.data(), new_parent, p_size * sizeof(int), hipMemcpyDeviceToHost), 
			"Failed to copy back new parent array");
		int j = 0;
		for(auto i : h_new_parent) {
			std::cout << "new_parent[" << j++ << "]= " << i << "\n";
		}
	}
}

