#include "hip/hip_runtime.h"
//---------------------------------------------------------------------
// Standard Libraries
//---------------------------------------------------------------------
#include <vector>
#include <chrono>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include <filesystem>

//---------------------------------------------------------------------
// CUDA Libraries
//---------------------------------------------------------------------
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

//---------------------------------------------------------------------
// CUDA Kernels
//---------------------------------------------------------------------
#include "cuda_bfs/cuda_csr.cuh"
#include "cuda_bfs/cuda_bfs.cuh"
#include "common/cuda_utility.cuh"
#include "common/Timer.hpp"

#include "connected_components/cc.cuh"

#include <hip/hip_runtime.h>
#include <iostream>
#include <hipcub/hipcub.hpp>

#include "dynamic_spanning_tree/update_ds.cuh"
#include "common/cuda_utility.cuh"

using namespace hipcub;

// #define DEBUG

CachingDeviceAllocator g_allocator_(true);  // Caching allocator for device memory

__device__ __forceinline__
long binary_search(uint64_t* array, long num_elements, uint64_t key) {
    long left = 0;
    long right = num_elements - 1;
    while (left <= right) {
        long mid = left + (right - left) / 2;
        if (array[mid] == key) {
            return mid; // Key found
        }
        if (array[mid] < key) {
            left = mid + 1;
        } else {
            right = mid - 1;
        }
    }
    return -1; // Key not found
}

__global__
void mark_delete_edges_kernel(
    uint64_t* d_edge_list,  // size <- numEdges
    long num_edges,
    uint64_t* d_edges_to_delete, // size <- delete_batch_size
    int delete_batch_size, 
    unsigned char* d_flags)     // size <- numEdges
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < delete_batch_size) {

        uint64_t t = d_edges_to_delete[tid];

        // t is the key, to be searched in the d_edge_list array
        long pos = binary_search(d_edge_list, num_edges, t);
        if(pos != -1) {
            d_flags[pos] = 0;
        }
    }
}

template <typename T>
void DisplayResults(T* arr, int num_items) {
    for(int i = 0; i < num_items; ++i) {
        printf("%llu ", (unsigned long long)arr[i]);
    }
    printf("\n");
}

void DisplayDeviceUint64Array_(uint64_t* d_arr, unsigned char* d_flags, int num_items) {
    // Allocate host memory for the copy
    uint64_t* h_arr = new uint64_t[num_items];
    
    // Copy data from device to host
    hipMemcpy(h_arr, d_arr, sizeof(uint64_t) * num_items, hipMemcpyDeviceToHost);
    
    unsigned char* flag_arr = new unsigned char[num_items];
    hipMemcpy(flag_arr, d_flags, sizeof(unsigned char) * num_items, hipMemcpyDeviceToHost);
    
    std::cout << "Device h_in Array: \n";
    for(int i = 0; i < num_items; ++i) {
        std::cout << h_arr[i] << " <-- " << static_cast<int>(flag_arr[i]) << "\n";
    }
    std::cout << std::endl;
    
    // Cleanup host memory
    delete[] h_arr;
    delete[] flag_arr;
}

void sort_array_uint64_t_(uint64_t* d_data, long num_items) {
    // Allocate temporary storage for sorting
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    auto start = std::chrono::high_resolution_clock::now();
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_data, d_data, num_items);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Run sorting operation
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_data, d_data, num_items);
    hipDeviceSynchronize();

    CUDA_CHECK(hipFree(d_temp_storage), "Failed to free d_temp_storage");
}

void select_flagged_(uint64_t* d_in, uint64_t* d_out, unsigned char* d_flags, long& num_items) {

    if(g_verbose) {
        DisplayDeviceUint64Array_(d_in, d_flags, num_items);
        // DisplayDeviceUCharArray(d_flags, num_items);
    }
    
    long     *d_num_selected_out   = NULL;
    g_allocator_.DeviceAllocate((void**)&d_num_selected_out, sizeof(long));

    // Allocate temporary storage
    void        *d_temp_storage = NULL;
    size_t      temp_storage_bytes = 0;

    DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out, d_num_selected_out, num_items);
    g_allocator_.DeviceAllocate(&d_temp_storage, temp_storage_bytes);

    // Run
    DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out, d_num_selected_out, num_items);

    long h_num;
    hipMemcpy(&h_num, d_num_selected_out, sizeof(long), hipMemcpyDeviceToHost);
    std::cout << "\nh_num: " <<  h_num << std::endl;
    num_items = h_num;
    // Copy output data back to host
    uint64_t* h_out = new uint64_t[num_items];
    hipMemcpy(h_out, d_out, sizeof(uint64_t) * num_items, hipMemcpyDeviceToHost);

    if(g_verbose) {
        // Print output data
        printf("\nOutput Data (h_out):\n");
        DisplayResults(h_out, h_num); // Print only the selected elements
    }

    CUDA_CHECK(hipFree(d_temp_storage), "Failed to free d_temp_storage");
    CUDA_CHECK(hipFree(d_num_selected_out), "Failed to free d_num_selected_out");

}

void update_edgelist_bfs(
    uint64_t* d_edge_list, uint64_t* d_updated_ed_list, 
    long& num_edges, 
    uint64_t* d_edges_to_delete, int delete_size) {

    // sort the input edges
    sort_array_uint64_t_(d_edge_list, num_edges);
    
    // init d_flag with true values
    unsigned char   *d_flags = NULL;
    std::vector<unsigned char> h_flags(num_edges, 1);
    CUDA_CHECK(hipMalloc((void**)&d_flags, sizeof(unsigned char) * num_edges), 
        "Failed to allocate memory for d_flags");

    CUDA_CHECK(hipMemcpy(d_flags, h_flags.data(), sizeof(unsigned char) * num_edges, hipMemcpyHostToDevice),
        "Failed to copy back d_flags");

    int numThreads = 1024;
    int numBlocks = (delete_size + numThreads - 1) / numThreads;

    // Launch kernel to mark batch edges for deletion in the actual edge_list
    mark_delete_edges_kernel<<<numThreads, numBlocks>>>(
        d_edge_list, 
        num_edges, 
        d_edges_to_delete, 
        delete_size, 
        d_flags
    );

    // now delete the edges from the graph array
    select_flagged_(d_edge_list, d_updated_ed_list, d_flags, num_edges);

    // if(g_verbose) {
    //     std::cout << "printing updated edgelist:\n";
    //     std::cout << "numEdges after delete batch: " << num_edges << "\n";
    //     print_device_edge_list(d_updated_ed_list, num_edges);
    // }
    CUDA_CHECK(hipFree(d_flags), "Failed to free d_flags");
}

__global__ 
void setParentLevelKernel(int* d_parent, int* d_level, int root) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_parent[root] = root;
        d_level[root] = 0;
    }
}

__global__ 
void simpleBFS( 
	int no_of_vertices, int level, 
    int* d_parents, int* d_levels, 
    long* d_offset, int* d_neighbour, 
    int* d_changed) {
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < no_of_vertices && d_levels[tid] == level) {
        int u = tid;
        for (long i = d_offset[u]; i < d_offset[u + 1]; i++) {
            int v = d_neighbour[i];
            if(d_levels[v] < 0) {
                d_levels[v] = level + 1;
                d_parents[v] = u;
                *d_changed = 1;
            }
        }
    }
}

void constructSpanningTree(
    int no_of_vertices, 
    long numEdges, 
    long* d_offset, 
    int* d_neighbours, 
    int* d_level, 
    int* d_parent, 
    int root) 
{

    #ifdef DEBUG
        g_verbose = true;
    #endif

    int level = 0;
    int totalThreads = 1024;
    int no_of_blocks = (no_of_vertices + totalThreads - 1) / totalThreads;
    
    int* d_changed;
    hipMallocManaged(&d_changed, sizeof(int));

    *d_changed= 1;

    setParentLevelKernel<<<1, 1>>>(d_parent, d_level, root);
    CUDA_CHECK(hipDeviceSynchronize(), "Failed to launch setParentLevelKernel.");

    while (*d_changed) {
        *d_changed = 0;
        
        simpleBFS<<<no_of_blocks, totalThreads>>>(
            no_of_vertices, 
            level, 
            d_parent, 
            d_level, 
            d_offset, 
            d_neighbours, 
            d_changed
        );
        CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize after simpleBFS");
        ++level;
    }

    std::cout << "Depth of tree: " << level << std::endl;

    CUDA_CHECK(hipFree(d_changed), "Failed to free d_changed");
}

// ====[ End of constructSpanningTree Code ]====


__global__
void get_original_edges(uint64_t* d_edgeList, int* original_u, int* original_v, long numEdges) {
	
	long tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < numEdges) { 
	   uint64_t t = d_edgeList[tid];
        original_u[tid] = (int)t & 0xFFFFFFFF;
        original_v[tid] = (int)(t >> 32);
    }
}

__global__
void print_original_edges(int* original_u, int* original_v, long numEdges) {
    long tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid == 0) {
        printf("Printing edgelist from bfs:\n");
        for(long i = 0; i < numEdges; ++i) {
            printf("edge[%ld]: (%d, %d)\n", i, original_u[i], original_v[i]);
        }
    }
}

void print_CSR(const std::vector<long>& vertices, const std::vector<int>& edges) {
    int numVertices = vertices.size() - 1;
    for (int i = 0; i < numVertices; ++i) {
        std::cout << "Vertex " << i << " is connected to: ";
        for (int j = vertices[i]; j < vertices[i + 1]; ++j) {
            std::cout << edges[j] << " ";
        }
        std::cout << "\n";
    }
}

void read_delete_batch(const std::string& delete_filename, std::vector<uint64_t>& edges_to_delete) {

    std::ifstream inputFile(delete_filename);
    if (!inputFile) {
        std::cerr << "Failed to open file: " << delete_filename << std::endl;
        return;
    }
    
    // n_edges: Number of edges to delete, including both tree and non-tree edges.
    int n_edges;
    inputFile >> n_edges;

    uint32_t u, v;
    edges_to_delete.resize(n_edges);
    
    std::cout << "Reading " << n_edges << " edges from the file." << std::endl;

    for (int i = 0; i < n_edges; ++i) {
        inputFile >> u >> v;
        if(u > v) {
            // Ensures u is always less than v for consistent edge representation
            std::swap(u, v);
        }

        edges_to_delete[i] = ((uint64_t)(u) << 32 | v);
    }

    if(g_verbose) {

        std::cout << "edges_to_delete array uint64_t:\n";

        for(auto i : edges_to_delete)
            std::cout << i <<" ";
        std::cout << std::endl;

        std::cout << "edges_to_delete array:\n";
        for(const auto &i : edges_to_delete)
            std::cout << (i >> 32) << " " << (i & 0xFFFFFFFF) << "\n";
        std::cout << std::endl;
    }
}

void cuda_BFS(graph& G, const std::string& delete_filename) {

    int numVert     =   G.numVert;
    long numEdges   =   G.numEdges / 2;
    
    // delete the edges
    std::vector<uint64_t> edges_to_delete;
    read_delete_batch(delete_filename, edges_to_delete);

    uint64_t* d_edge_list = nullptr;
    uint64_t* d_updated_ed_list = nullptr;
    uint64_t* d_edges_to_delete = nullptr;

    size_t delete_size = edges_to_delete.size() * sizeof(uint64_t);
    size_t edge_list_size = G.edge_list.size() * sizeof(uint64_t);

    // if(g_verbose) {
    //     std::cout << "Edge list from cuda_BFS:\n";
    //     for(auto i : G.edge_list) 
    //         std::cout << (i >> 32) <<" " << (i & 0xFFFFFFFF) << " <- " << i << "\n";
    //     std::cout << std::endl;
    // }
    
    CUDA_CHECK(hipMalloc(&d_edge_list, edge_list_size), "Failed to allocate memory for input edge list");
    CUDA_CHECK(hipMalloc(&d_updated_ed_list, edge_list_size), "Failed to allocate memory for input edge list");
    CUDA_CHECK(hipMalloc(&d_edges_to_delete, delete_size), "Failed to allocate memory for edges to delete");

    CUDA_CHECK(hipMemcpy(d_edge_list, G.edge_list.data(), edge_list_size, hipMemcpyHostToDevice), "Failed to copy edge list to device");
    CUDA_CHECK(hipMemcpy(d_edges_to_delete, edges_to_delete.data(), delete_size, hipMemcpyHostToDevice), "Failed to copy edges to delete to device");

	int* original_u;  // single edges
	int* original_v;

	hipMalloc((void **)&original_u, numEdges * sizeof(int));
    hipMalloc((void **)&original_v, numEdges * sizeof(int));

    update_edgelist_bfs(d_edge_list, d_updated_ed_list, numEdges, d_edges_to_delete, delete_size);

	long E = 2 * numEdges; // Two times the original edges count (0,1) and (1,0).
	
    // step 1: Create duplicates
	int* u_arr_buf;
	int* v_arr_buf;
	int* u_arr_alt_buf;
	int* v_arr_alt_buf;

	// Allocate memory for duplicates
    hipMalloc((void **)&u_arr_buf, E * sizeof(int));
    hipMalloc((void **)&v_arr_buf, E * sizeof(int));
    hipMalloc((void **)&u_arr_alt_buf, E * sizeof(int));
    hipMalloc((void **)&v_arr_alt_buf, E * sizeof(int));

    long* d_vertices;
	hipMalloc((void **)&d_vertices, (numVert + 1) * sizeof(long));

	int *d_parent;
	int *d_level;

	hipMalloc((void **)&d_parent,  numVert * sizeof(int));
    hipMalloc((void **)&d_level,   numVert * sizeof(int));

    CUDA_CHECK(hipMemset(d_level, -1, numVert * sizeof(int)), "Failed to initialize level array.");

    int totalThreads = 1024;
    int numBlocks = (numEdges + totalThreads - 1) / totalThreads;

    // std::cout << "Timer started" << std::endl;

    get_original_edges<<<numBlocks, totalThreads>>>(d_updated_ed_list, original_u, original_v, numEdges);
    CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize get_original_edges");
    
    // validate once, if the tree is connected or not after deleting edges.
    
    cc(original_u, original_v, numVert, numEdges);

    Timer myTimer;
    myTimer.start();
	create_duplicate(original_u, original_v, u_arr_buf, v_arr_buf, numEdges);
	// Step [i]: alternate buffers for sorting operation
	// Create DoubleBuffers
	hipcub::DoubleBuffer<int> d_u_arr(u_arr_buf, u_arr_alt_buf);
	hipcub::DoubleBuffer<int> d_v_arr(v_arr_buf, v_arr_alt_buf);

	// Output: 
	// Vertices array			-> d_vertices <- type: long;
	// Neighbour/edges array	-> d_v_arr.Current() <- type: int;

	gpu_csr(d_u_arr, d_v_arr, E, numVert, d_vertices);
	// CSR creation ends here

    // if(g_verbose) {
    //     // print gpu_CSR
    //     size_t size = E * sizeof(int);
    //     std::vector<long> host_vert(numVert + 1);
    //     std::vector<int> host_edges(E);
    //     CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize stream before hipMemcpyAsync in gpu_csr");
    //     // Use hipMemcpyAsync with the stream for asynchronous memory copy
    //     CUDA_CHECK(hipMemcpy(host_vert.data(), d_vertices, (numVert + 1) * sizeof(long), hipMemcpyDeviceToHost), 
    //                 "Failed to copy back vertices array.");
    //     CUDA_CHECK(hipMemcpy(host_edges.data(), d_v_arr.Current(), size, hipMemcpyDeviceToHost), 
    //                 "Failed to copy back edges array.");

    //     print_CSR(host_vert, host_edges);
    // }

    // std::cout << "d_level array from cuda_BFS:\n";
    // print_device_array(d_level, numVert);

	int root = 0;
	// Step 1: Construct a rooted spanning tree
	constructSpanningTree(
		numVert, 
		E, 
		d_vertices, 
		d_v_arr.Current(), 
		d_level, 
		d_parent, 
		root);

    std::cout << "Total elapsed time for cudaBFS: " << myTimer.getElapsedMilliseconds() << " ms" << std::endl;

    // call adam_polak bfs
    adam_polak_bfs(numVert, E, d_vertices, d_v_arr.Current());    

	// Cleanup
    hipFree(original_u);
    hipFree(original_v);
    hipFree(u_arr_buf);
    hipFree(v_arr_buf);
    hipFree(u_arr_alt_buf);
    hipFree(v_arr_alt_buf);
	hipFree(d_vertices);
	hipFree(d_parent);
	hipFree(d_level);
    hipFree(d_edge_list);
    hipFree(d_updated_ed_list);
    hipFree(d_edges_to_delete);
}