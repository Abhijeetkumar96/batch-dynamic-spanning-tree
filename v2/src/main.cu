#include <iostream>
#include <vector>
#include <fstream>
#include <chrono>

#include "graph.hpp"
#include "bfs.hpp"

#include "dynamic_tree_util.cuh"
#include "dynamic_tree.cuh"
#include "cuda_utility.cuh"
#include "euler_tour.cuh"

#define DEBUG

int main(int argc, char* argv[]) {
    if(argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <path_to_graph> <delete_batch>" << std::endl;
        return EXIT_FAILURE;
    }
    CUDA_CHECK(hipSetDevice(2), "Failed to set device");
    
    std::string filename = argv[1];
    std::string delete_filename = argv[2];
    
    graph G(filename);

    std::cout << "numVertices : " << G.numVert << ", numEdges : " << G.numEdges << std::endl;

    std::vector<int> parent(G.numVert);
    std::vector<int> roots;
    
    std::cout << "\t\tBFS Started...\n";
    int numComp = bfs(G.vertices, G.edges, parent, roots);

    std::cout <<"Number of components in the graph : " << numComp << std::endl;

    #ifdef DEBUG
        // G.print_CSR();
        // G.print_list();
        std::cout << "Parent array:\n";
        host_print(parent);
    #endif
        
    // calculate the eulerian tour
    EulerianTour euler_tour(G.numVert);

    dynamic_tree_manager tree_ds;
    tree_ds.read_delete_batch(delete_filename);
    tree_ds.mem_alloc(parent, G.edge_list);
    tree_ds.update_existing_ds();
        
    std::cout << "The edge list has been updated.\n";

    repair_spanning_tree(roots, tree_ds, euler_tour);

    return EXIT_SUCCESS; 
}
