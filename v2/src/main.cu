#include <iostream>
#include <vector>
#include <fstream>
#include <chrono>

#include "serial_rst/spanning_tree.hpp"

#include "common/graph.hpp"
#include "common/cuda_utility.cuh"

#include "dynamic_spanning_tree/dynamic_tree_util.cuh"
#include "dynamic_spanning_tree/dynamic_tree.cuh"
#include "dynamic_spanning_tree/euler_tour.cuh"

#include "cuda_bfs/cuda_bfs.cuh"

bool    checker             = false;
bool    g_verbose           = false;  // Whether to display i/o to console
long    maxThreadsPerBlock  = 0;

int validate(const int*, int);

int main(int argc, char* argv[]) {
    if(argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <path_to_graph> <delete_batch>" << std::endl;
        return EXIT_FAILURE;
    }

    cuda_init(2);
    
    std::string filename = argv[1];
    std::string delete_filename = argv[2];
    
    graph G(filename);

    std::cout << "numVertices : " << G.numVert << ", numEdges : " << G.numEdges << std::endl;

    // baseline_1: bfs (simple + Adam_Polak)
    cuda_BFS(G, delete_filename);

    g_verbose = false;
    
    std::vector<int> parent(G.numVert);
    std::vector<int> roots;
    
    std::cout << "\t\tBFS Started...\n";
    int numComp = bfs(G.vertices, G.edges, parent, roots);

    std::cout <<"Number of components in the graph : " << numComp << std::endl;

    if(g_verbose) {
        // G.print_CSR();
        G.print_list();
        std::cout << "\nParent array from main function:\n";
        int j = 0;
        for(auto i : parent) 
            std::cout << "parent[" << j++ << "] = " << i << std::endl;
        std::cout << std::endl;
    }
        
    // calculate the eulerian tour
    EulerianTour euler_tour(G.numVert);

    dynamic_tree_manager tree_ds(parent, delete_filename, G.edge_list);

    // if(g_verbose) {
    //     std::cout << "updated edgelist from main:\n";
    //     print_device_edge_list(tree_ds.d_updated_edge_list, tree_ds.num_edges);

    //     std::cout << "The edge list has been updated.\n";
    // }

    repair_spanning_tree(roots, tree_ds, euler_tour);

    // validate the output
    int* new_parent = tree_ds.new_parent;

    int temp = validate(new_parent, G.numVert);
    std::cout << "numComp after edge deletion: " << temp << std::endl;

    return EXIT_SUCCESS; 
}

int validate(const int* parent, int n) {

    std::cout << "Executing validate part.\n";
    int* new_parent;
    CUDA_CHECK(hipMalloc(&new_parent, n * sizeof(int)), "Failed to allocate memory for new_parent");
    CUDA_CHECK(hipMemcpy(new_parent, parent, n * sizeof(int), hipMemcpyDeviceToDevice), "Failed to copy parent array");

    int* d_out;
    CUDA_CHECK(hipMalloc(&d_out, n * sizeof(int)), "Failed to allocate memory for d_out");

    std::cout << "Doing pointer_jumping.\n";
    pointer_jumping(new_parent, n);
    int result;
    find_unique(new_parent, d_out, n, result);

    // CUDA_CHECK(hipFree(new_parent), "Failed to free new_parent");
    // CUDA_CHECK(hipFree(d_out), "Failed to free d_out");

    return result;
}