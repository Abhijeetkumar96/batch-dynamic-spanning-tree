#include "hip/hip_runtime.h"
#include "grafting.h"
#include "utility.h"

__global__
void DetermineWinners(int *u_arr, int *v_arr, int *rep, int *winner, int edges, int *d_flag) {
	
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < edges) {
		// for i from 1 to n:
		//for each neighbor j of vertex i:
		//Assuming u as vertex i and v as all neighbours of u
		int u = u_arr[tid];
		int v = v_arr[tid];

		int rep_u = rep[u], rep_v = rep[v];

		if(rep_u != rep_v) {
			winner[max(rep_u,rep_v)] = tid;
			*d_flag = 1;
		}
	}
}

__global__ 
void UpdateLabels(int *u_arr, int *v_arr, int *rep, int *winner, int edges, int *marked_parent, int *onPath)
{

  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < edges)
  {
    int u = u_arr[tid];
    int v = v_arr[tid];

	int rep_u = rep[u], rep_v = rep[v];

	if(rep_u != rep_v && winner[max(rep_u,rep_v)] == tid)
	{
		if(rep_u > rep_v)
		{
			marked_parent[u] = v;
			onPath[u] = 1;
		}
		else
		{
			marked_parent[v] = u;
			onPath[v] = 1;	
		}

	}
  }
}

void Graft(
	int vertices,
	int edges,
	int *d_u_ptr,
	int *d_v_ptr,
	int *d_ptr,
	int *d_winner_ptr,
	int *d_marked_parent,
	int *d_OnPath,
	int *d_flag
)
{
		int numThreads = 1024;
		// int numBlocks_n = (vertices + numThreads - 1) / numThreads;
		int numBlocks_e = (edges + numThreads - 1) / numThreads;

		// Step 2.1: Determine potential winners for each vertex
		DetermineWinners<<<numBlocks_e, numThreads>>> (d_u_ptr, d_v_ptr, d_ptr, d_winner_ptr, edges, d_flag);
		hipDeviceSynchronize();

		// Step 2.2: Update labels based on winners and mark parents
    	UpdateLabels<<<numBlocks_e, numThreads>>>(d_u_ptr, d_v_ptr, d_ptr, d_winner_ptr, edges, d_marked_parent, d_OnPath);
    	hipDeviceSynchronize();	
}