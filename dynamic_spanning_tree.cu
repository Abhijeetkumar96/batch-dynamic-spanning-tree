#include "hip/hip_runtime.h"
#include "cuda_utility.cuh"
#include "euler_tour.cuh"
#include "dynamic_tree.cuh"

__device__ __forceinline__
long binary_search(uint64_t* array, long num_elements, uint64_t key) {
    long left = 0;
    long right = num_elements - 1;
    while (left <= right) {
        long mid = left + (right - left) / 2;
        if (array[mid] == key) {
            return mid; // Key found
        }
        if (array[mid] < key) {
            left = mid + 1;
        } else {
            right = mid - 1;
        }
    }
    return -1; // Key not found
}

__global__
void delete_edges_kernel(
    int* d_parent,          // size <- numNodes
    uint64_t* d_edge_list,  // size <- numEdges
    long num_edges,         
    uint64_t* d_edges_to_delete, // size <- delete_batch_size
    int delete_batch_size, 
    unsigned char* flags)       // size <- numEdges
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < delete_batch_size) {
        uint32_t u, v;
        uint64_t t = d_edges_to_delete[tid];
        uint32_t u = (uint32_t)(t >> 32);
        uint32_t v = (uint32_t)(t & 0xFFFFFFFF); 

        // delete tree edges
        if(u == d_parent[v]) {
            d_parent[v] = v;
        }
        else if(v == parent[u]) {
            parent[u] = u;
        }

        else {
            // t is the key, to be searched in the d_edge_list array
            long pos = binary_search(d_edge_list, num_edges, t);
            if(pos != -1) {
                d_flag[pos] = 0;
            }
        }
    }
}

void sort_array_uint64_t(uint64_t* d_data, long num_items) {
    // Allocate temporary storage for sorting
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    auto start = std::chrono::high_resolution_clock::now();
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_data, d_data, num_items);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Run sorting operation
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_data, d_data, num_items);
    hipDeviceSynchronize();
}

void select_flagged(uint64_t* d_in, uint64_t* d_out, unsigned char* d_flags, long& num_items) {
    // Allocate device output array and num selected
    long *d_num_selected_out   = NULL;
    
    CUDA_CHECK(hipMalloc((void**)&d_out, sizeof(uint64_t) * num_items), "Failed to allocate memory for d_out");
    CUDA_CHECK(hipMalloc((void**)&d_num_selected_out, sizeof(long)), "Failed to allocate memory for d_num_selected_out");

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;
    
    DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out, d_num_selected_out, num_items);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Run
    DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out, d_num_selected_out, num_items);

    CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize after hipcub::Flagged");

    CUDA_CHECK(hipMemcpyAsync(&num_items, d_num_selected_out, sizeof(long), hipMemcpyDeviceToHost),"Failed to copy back d_num_selected_out");
    CUDA_CHECK(hipDeviceSynchronize(), "Failed to copy back num_items");
}

void update_existing_ds(
    int* d_parent, int* d_rep, int num_vert, 
    uint64_t* d_edge_list, long num_edges, 
    uint64_t* d_edges_to_delete, int delete_size) {

    // sort the input edges
    sort_array_uint64_t(d_edge_list, num_edges);
    
    // init d_flag with true values
    unsigned char   *d_flags = NULL;
    std::vector<unsigned char> h_flags(num_edges, 1);
    CUDA_CHECK(hipMalloc((void**)&d_flags, sizeof(unsigned char) * num_edges), 
        "Failed to allocate memory for d_flags");

    delete_edges_kernel<<<<<<numThreads, numBlocks>>>(
        d_parent, 
        d_edge_list, 
        num_edges, 
        d_edges_to_delete, 
        delete_size, 
        d_flags
    );

    CUDA_CHECK(hipDeviceSynchronize(), "Failed to delete edges");

    CUDA_CHECK(
        hipMemcpyAsync(
            d_rep, 
            d_parent, 
            num_vert * sizeof(int), 
            hipMemcpyDeviceToDevice,
            ), 
        "Failed to copy parent from device to device"
    );

    uint64_t* d_updated_ed_list = nullptr;

    // Initialize device input
    hipMemcpy(d_in, h_in, sizeof(int) * num_items, hipMemcpyHostToDevice);
    hipMemcpy(d_flags, h_flags, sizeof(unsigned char) * num_items, hipMemcpyHostToDevice);
    
    // now delete the edges from the parent array
    select_flagged(d_edge_list, d_updated_ed_list, d_flags, num_edges);
}
    
void dynamic_tree_manager::mem_alloc(const std::vector<int>& parent, const std::vector<uint64_t>& edge_list) {

	num_vert = parent.size();
    num_edges = edge_list.size();
    
    size_t size = parent.size() * sizeof(int);
    size_t delete_size = edges_to_delete.size() * sizeof(uint64_t);
    size_t num_edges = edge_list.size() * sizeof(uint64_t);
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_parent, size), "Failed to allocate memory for d_parent");
    CUDA_CHECK(hipMalloc(&d_rep, size), "Failed to allocate memory for d_rep");
    CUDA_CHECK(hipMalloc(&d_unique_rep, size), "Failed to allocate memory for d_unique_rep");
    CUDA_CHECK(hipMalloc(&d_edges_to_delete, delete_size), "Failed to allocate memory for edges to delete");
    CUDA_CHECK(hipMalloc(&d_edge_list, num_edges), "Failed to allocate memory for input edge list");

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_parent, parent.data(), size, hipMemcpyHostToDevice), "Failed to copy d_parent to device");
    CUDA_CHECK(hipMemcpy(d_edges_to_delete, edges_to_delete.data(), delete_size, hipMemcpyHostToDevice), "Failed to copy edges to delete to device");
    CUDA_CHECK(hipMemcpy(d_edge_list, edge_list.data(), num_edges, hipMemcpyHostToDevice), "Failed to copy edge list to device");
    
    // Create a hash table on the device
    pHashTable = create_hashtable();
}

dynamic_tree_manager::~dynamic_tree_manager() {
    hipFree(d_parent);
    hipFree(d_rep);
    hipFree(d_unique_rep);
    hipFree(d_edges_to_delete);
    hipFree(d_edge_list);
    destroy_hashtable(pHashTable);
}

keyValues* create_hashtable() {
    keyValues* hashtable;
    CUDA_CHECK(hipMalloc(&hashtable, sizeof(keyValues) * kHashTableCapacity), "Failed to allocate hashtable");
    CUDA_CHECK(hipMemset(hashtable, 0xff, sizeof(keyValues) * kHashTableCapacity), "Failed to initialize hashtable");
    return hashtable;
}

void destroy_hashtable(keyValues* pHashTable) {
    CUDA_CHECK(hipFree(pHashTable), "Failed to free hashtable");
}

void dynamic_tree_manager::read_delete_batch(const std::string& delete_filename) {

    std::ifstream inputFile(delete_filename);
    if (!inputFile) {
        std::cerr << "Failed to open file: " << delete_filename << std::endl;
        return;
    }
    
    // n_edges: Number of edges to delete, including both tree and non-tree edges.
    int n_edges;
    inputFile >> n_edges;
    delete_batch_size = n_edges;
    uint32_t u, v;
    edges_to_delete.resize(n_edges);
    
    std::cout << "Reading " << n_edges << " edges from the file." << std::endl;

    for (int i = 0; i < n_edges; ++i) {
        inputFile >> u >> v;
        if(u > v) {
            // Ensures u is always less than v for consistent edge representation
            std::swap(u, v);
        }
        edges_to_delete[i] = ((uint64_t)(u) << 32 | v);
    }
}

void dynamic_tree_manager::update_existing_ds() {
	update_existing_ds(d_parent, d_rep, num_vert, d_edge_list, num_edges, d_edges_to_delete, delete_batch_size);
}

void repair_spanning_tree() {
	
	cal_first_last(roots[0], d_parent, euler_tour);

}



// ====[ End of update ds Code ]====