#include "hip/hip_runtime.h"
#include <set>
#include <hip/hip_runtime.h>

#include "super_graph/super_graph.cuh"
#include "dynamic_spanning_tree/euler_tour.cuh"
#include "common/cuda_utility.cuh"
#include "common/Timer.hpp"
#include "dynamic_spanning_tree/dynamic_tree.cuh"
#include "PR-RST/pr_rst_util.cuh"
#include "dynamic_spanning_tree/path_reversal.cuh"

// #define DEBUG
// #define CHECKER

__global__
void update_rep_map(int* d_unique_rep, int* d_rep_map, int unique_rep_count) {
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < unique_rep_count) {
        // printf("tid: %d, unique_rep: %d, rep_map pos: %d\n", tid, d_unique_rep[tid], tid);
    	d_rep_map[d_unique_rep[tid]] = tid;
    }
}

void repair_spanning_tree(const std::vector<int>& roots, dynamic_tree_manager& tree_ds, EulerianTour& euler_tour) {

	int* d_rep 			= 	tree_ds.d_parent;
	int* d_unique_rep 	= 	tree_ds.d_unique_rep;
	int* d_rep_map 		= 	tree_ds.d_rep_map;
	int num_vert 		= 	tree_ds.num_vert;
	
	// Timer myTimer;
    // myTimer.start();
    // std::cout << "Timer started" << std::endl;

	#ifdef DEBUG
		std::cout << "parent array after deleting edges:\n";
		print_device_array(d_rep, num_vert);
	#endif

	auto start = std::chrono::high_resolution_clock::now();
	// 1. find eulerian tour
	cal_first_last(roots[0], tree_ds.d_org_parent, euler_tour);

	// if num of components in the forest > 1, then call multi-component eulerian_tour
	// else call normal eulerian_tour

	// if(num_comps > 1)
	// 	cal_first_last(roots[0], tree_ds.d_org_parent, euler_tour);
	// else
	// 	cal_first_last(roots[0], tree_ds.d_org_parent, euler_tour);

	auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration<double, std::milli>(stop - start).count();

    add_function_time("Eulerian Tour", duration);

	CUDA_CHECK(hipMemcpy(tree_ds.new_parent, tree_ds.d_parent, num_vert * sizeof(int), hipMemcpyDeviceToDevice), 
        "Failed to copy d_parent to device");

	start = std::chrono::high_resolution_clock::now();
	// 2. Do pointer jumping over parent array to update representative array.
	pointer_jumping(d_rep, num_vert);

	stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration<double, std::milli>(stop - start).count();

    add_function_time("Pointer Jumping", duration);
	
	#ifdef CHECKER
		std::vector<int> h_rep(tree_ds.num_vert);
		CUDA_CHECK(hipMemcpy(h_rep.data(), d_rep, sizeof(int) * tree_ds.num_vert, hipMemcpyDeviceToHost),
		"Failed to copy d_rep array to host");

		bool result = compare_arrays(h_rep.data(), tree_ds.parent_array, tree_ds.num_vert);

    	std::cout << "Comparison of cpu p_jump and gpu p_jump: " << (result ? "Equal" : "Not Equal") << std::endl;

	    std::set<int> unique_elements(h_rep.begin(), h_rep.end());
	    std::cout << "Unique representatives after deleting edges: " << unique_elements.size() << "\n";
    	
    	// std::cout << "Unique representatives in sorted order: \n";
    	// for (int element : unique_elements) {
        // 	std::cout << element << " ";
    	// }
    	// std::cout << std::endl;
    #endif

	#ifdef DEBUG
		std::cout << "After doing pointer_jumping:\n";
		print_device_array(d_rep, num_vert);
	#endif
	
	// 3. find unique in the d_rep array
	int unique_rep_count = tree_ds.unique_rep_count;

	#ifdef DEBUG
		std::cout << "unique_rep_count: " << unique_rep_count << std::endl;
		std::cout << "d_unique_rep array:\n";
		print_device_array(d_unique_rep, unique_rep_count);
	#endif

	int numThreads = 1024;
	int numBlocks = (unique_rep_count + numThreads - 1) / numThreads;
	
	start = std::chrono::high_resolution_clock::now();
	// update rep_map
	update_rep_map<<<numBlocks, numThreads>>>(d_unique_rep, d_rep_map, unique_rep_count);
	CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize after update_rep_map");

	stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration<double, std::milli>(stop - start).count();

    add_function_time("Update rep array", duration);
	
	#ifdef DEBUG	
		std::cout << "d_rep_map array:\n";
		print_device_array(d_rep_map, num_vert);
	#endif
    
    // myTimer.pause();
	PR_RST resource_mag(unique_rep_count);
	// myTimer.resume();
	// weed out self loops and duplicates and get the replacement edges

	get_replacement_edges(tree_ds, resource_mag, unique_rep_count);

	int* unique_super_graph_edges = tree_ds.super_graph_edges;

	if(*unique_super_graph_edges < 1) {
        // std::cerr << "No cross edges found to connect the graphs.\n";
        return;
    }

    start = std::chrono::high_resolution_clock::now();
	path_reversal(tree_ds, euler_tour, resource_mag, unique_rep_count);
	stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration<double, std::milli>(stop - start).count();

    add_function_time("Path Reversal", duration);

    // std::cout << "Total elapsed time for dynamic_spanning_tree repair: " << myTimer.getElapsedMilliseconds() << " ms" << std::endl;

    // validation
    // do pointer_jumping and is_unique
}